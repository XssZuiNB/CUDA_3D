#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <stdint.h>

#include <memory>
#include <stdexcept>

template <typename T>
std::shared_ptr<T> make_device_copy(T obj) {
  T* d_data;
  auto res = hipMalloc(&d_data, sizeof(T));
  if (res != hipSuccess)
    throw std::runtime_error("hipMalloc failed status: " + res);
  hipMemcpy(d_data, &obj, sizeof(T), hipMemcpyHostToDevice);
  return std::shared_ptr<T>(d_data, [](T* data) { hipFree(data); });
}

template <typename T>
std::shared_ptr<T> alloc_dev(int elements) {
  T* d_data;
  auto res = hipMalloc(&d_data, sizeof(T) * elements);
  if (res != hipSuccess)
    throw std::runtime_error("hipMalloc failed status: " + res);
  return std::shared_ptr<T>(d_data, [](T* p) { hipFree(p); });
}
