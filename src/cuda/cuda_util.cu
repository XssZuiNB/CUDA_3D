#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <stdexcept>

#include "cuda_util.cuh"

/*
template <typename T>
std::shared_ptr<T> make_device_copy(T obj) {
  T* d_data;
  auto res = hipMalloc(&d_data, sizeof(T));

  if (res != hipSuccess)
    throw std::runtime_error("hipMalloc failed status: " + res);

  hipMemcpy(d_data, &obj, sizeof(T), hipMemcpyHostToDevice);

  return std::shared_ptr<T>(d_data, [](T* data) { hipFree(data); });
}
*/
template <typename T> bool alloc_dev(std::shared_ptr<T> &cuda_ptr, int elements)
{
    auto err = ::hipSuccess;

    T *d_data;
    if (err = hipMalloc(&d_data, sizeof(T) * elements) != ::hipSuccess)
        return false;

    cuda_ptr = std::shared_ptr<T>(d_data, [](T *p) { hipFree(p); });

    return true;
}
