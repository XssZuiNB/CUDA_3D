#include "cuda_container/cuda_container.hpp"
#include "cuda_container/cuda_frame.cuh"
#include "util/cuda_util.cuh"

#include <hip/hip_runtime_api.h>

namespace gca
{
cuda_depth_frame::cuda_depth_frame(uint32_t width, uint32_t height)
    : __m_impl(new cuda_depth_frame_impl(width, height))
{
}

cuda_depth_frame::cuda_depth_frame(const uint16_t *frame, uint32_t width, uint32_t height)
    : __m_impl(new cuda_depth_frame_impl(frame, width, height))
{
}

cuda_depth_frame::cuda_depth_frame(const cuda_depth_frame &other)
    : __m_impl(new cuda_depth_frame_impl(*(other.__m_impl)))
{
}

cuda_depth_frame::cuda_depth_frame(cuda_depth_frame &&other) noexcept
    : __m_impl(other.__m_impl)
{
    other.__m_impl = nullptr;
}

cuda_depth_frame &cuda_depth_frame::operator=(const cuda_depth_frame &other)
{
    if (this != &other)
    {
        *__m_impl = *(other.__m_impl);
    }
    return *this;
}

cuda_depth_frame &cuda_depth_frame::operator=(cuda_depth_frame &&other) noexcept
{
    if (this != &other)
    {
        if (__m_impl)
        {
            delete __m_impl;
        }

        __m_impl = other.__m_impl;
        other.__m_impl = nullptr;
    }
    return *this;
}

uint32_t cuda_depth_frame::get_depth_frame_width() const
{
    return __m_impl->get_frame_width();
}

uint32_t cuda_depth_frame::get_depth_frame_height() const
{
    return __m_impl->get_frame_height();
}

const thrust::device_vector<uint16_t> &cuda_depth_frame::get_depth_frame_vec() const
{
    return __m_impl->get_frame_vec();
}

void cuda_depth_frame::upload(const uint16_t *src, uint32_t width, uint32_t height)
{
    __m_impl->upload(src, width, height);
}

void cuda_depth_frame::clear()
{
    __m_impl->clear();
}

cuda_depth_frame::~cuda_depth_frame()
{
    if (__m_impl)
    {
        delete __m_impl;
    }
    __m_impl = nullptr;
}
} // namespace gca
