#include "hip/hip_runtime.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include <stdint.h>
#include <stdio.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sort.h>

#include "camera/camera_param.hpp"
#include "cuda_container/cuda_container.hpp"
#include "geometry/cuda_point_cloud_factory.cuh"
#include "geometry/geometry_util.cuh"
#include "geometry/type.hpp"
#include "util/cuda_util.cuh"

namespace gca
{
/************************************ CUDA bilateral_filter  ************************************/

__forceinline__ __device__ static float __gaussian(float x, float sigma)
{
    return exp(-(x * x) / (2 * sigma * sigma));
}

__device__ static float __bilateral_filter(const uint16_t *input, uint32_t input_width,
                                           uint32_t input_height, int index_x, int index_y,
                                           int filter_radius, float sigma_space, float sigma_depth)
{
    float sum_weight = 0.0f;
    float sum = 0.0f;

#pragma unroll
    for (int dy = -filter_radius; dy <= filter_radius; ++dy)
    {
#pragma unroll
        for (int dx = -filter_radius; dx <= filter_radius; ++dx)
        {
            int nx = index_x + dx;
            int ny = index_y + dy;

            if (nx >= 0 && nx < input_width && ny >= 0 && ny < input_height)
            {
                float weight = __gaussian(sqrtf(dx * dx + dy * dy), sigma_space) *
                               __gaussian(abs(input[index_y * input_width + index_x] -
                                              input[ny * input_width + nx]),
                                          sigma_depth);

                sum_weight += weight;
                sum += weight * input[ny * input_width + nx];
            }
        }
    }

    return sum / sum_weight;
}

/****************** Create point cloud from rgbd, include invalid point remove ******************/

__forceinline__ __device__ static void __transform_point_to_point(float to_point[3],
                                                                  const float from_point[3],
                                                                  const gca::extrinsics &extrin)
{
    to_point[0] = extrin.rotation[0] * from_point[0] + extrin.rotation[3] * from_point[1] +
                  extrin.rotation[6] * from_point[2] + extrin.translation[0];
    to_point[1] = extrin.rotation[1] * from_point[0] + extrin.rotation[4] * from_point[1] +
                  extrin.rotation[7] * from_point[2] + extrin.translation[1];
    to_point[2] = extrin.rotation[2] * from_point[0] + extrin.rotation[5] * from_point[1] +
                  extrin.rotation[8] * from_point[2] + extrin.translation[2];
}

__forceinline__ __device__ static void __depth_uv_to_xyz(const float uv[2], const float depth,
                                                         float xyz[3],
                                                         const gca::intrinsics &depth_intrin)
{
    auto z = depth;
    xyz[2] = z;
    xyz[0] = (uv[0] - depth_intrin.cx) * z / depth_intrin.fx;
    xyz[1] = (uv[1] - depth_intrin.cy) * z / depth_intrin.fy;
}

__forceinline__ __device__ static void __xyz_to_color_uv(const float xyz[3], float uv[2],
                                                         const gca::intrinsics &color_intrin)
{
    uv[0] = (xyz[0] * color_intrin.fx / xyz[2]) + color_intrin.cx;
    uv[1] = (xyz[1] * color_intrin.fy / xyz[2]) + color_intrin.cy;
}

__global__ static void __kernel_make_pointcloud_Z16_BGR8(
    gca::point_t *point_set_out, const uint32_t width, const uint32_t height,
    const uint16_t *depth_frame_data, const uint8_t *color_frame_data,
    const gca::intrinsics *depth_intrin_ptr, const gca::intrinsics *color_intrin_ptr,
    const gca::extrinsics *depth_to_color_extrin_ptr, const float depth_scale, float threshold_min,
    float threshold_max, bool if_bilateral_filter = false)
{
    __shared__ gca::intrinsics depth_intrin_shared;
    __shared__ gca::intrinsics color_intrin_shared;
    __shared__ gca::extrinsics depth_to_color_extrin_shared;

    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        depth_intrin_shared = *depth_intrin_ptr;
        color_intrin_shared = *color_intrin_ptr;
        depth_to_color_extrin_shared = *depth_to_color_extrin_ptr;
    }

    __syncthreads();

    int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
    int depth_y = blockIdx.y * blockDim.y + threadIdx.y;
    int depth_pixel_index = depth_y * width + depth_x;

    // Shared memory or texture memory loading of depth_frame_data and color_frame_data

    if (depth_x >= 0 && depth_x < width && depth_y >= 0 && depth_y < height)
    {
        float depth_value;
        // Extract depth value
        if (if_bilateral_filter)
            depth_value =
                __bilateral_filter(depth_frame_data, width, height, depth_x, depth_y, 7, 1, 50) *
                depth_scale;
        else
            depth_value = depth_frame_data[depth_pixel_index] * depth_scale;

        if (depth_value <= 0.0 || depth_value < threshold_min || depth_value > threshold_max)
        {
            point_set_out[depth_pixel_index].property = gca::point_property::invalid;
            return;
        }

        // Calculate depth_uv and depth_xyz
        float depth_uv[2] = {depth_x - 0.5f, depth_y - 0.5f};
        float depth_xyz[3];
        __depth_uv_to_xyz(depth_uv, depth_value, depth_xyz, depth_intrin_shared);

        // Calculate color_xyz
        float color_xyz[3];
        __transform_point_to_point(color_xyz, depth_xyz, depth_to_color_extrin_shared);

        // Calculate color_uv
        float color_uv[2];
        __xyz_to_color_uv(color_xyz, color_uv, color_intrin_shared);

        const int target_x = static_cast<int>(color_uv[0] + 0.5f);
        const int target_y = static_cast<int>(color_uv[1] + 0.5f);

        if (target_x >= 0 && target_x < width && target_y >= 0 && target_y < height)
        {
            gca::point_t p;

            p.coordinates.x = depth_xyz[0];
            p.coordinates.y = depth_xyz[1];
            p.coordinates.z = depth_xyz[2];

            const int color_index = 3 * (target_y * width + target_x);
            p.b = color_frame_data[color_index + 0];
            p.g = color_frame_data[color_index + 1];
            p.r = color_frame_data[color_index + 2];

            p.property = gca::point_property::inactive;

            point_set_out[depth_pixel_index] = p;
        }
        else
        {
            point_set_out[depth_pixel_index].property = gca::point_property::invalid;
            return;
        }
    }
}

/******************************* Functor check if a point is valid ******************************/
struct check_is_valid_point_functor
{
    __host__ __device__ __forceinline__ bool operator()(gca::point_t p)
    {
        return p.property != gca::point_property::invalid;
    }
};

__forceinline__ static void remove_invalid_points(thrust::device_vector<gca::point_t> &result)
{
    thrust::device_vector<gca::point_t> temp(result.size());
    auto new_size = thrust::copy_if(result.begin(), result.end(), temp.begin(),
                                    check_is_valid_point_functor()) -
                    temp.begin();
    temp.resize(new_size);

    result.swap(temp);
}

/************************* For Debug, not be used in point cloud class **************************/
bool cuda_make_point_cloud(std::vector<gca::point_t> &result,
                           const gca::cuda_depth_frame &cuda_depth_container,
                           const gca::cuda_color_frame &cuda_color_container,
                           const gca::cuda_camera_param &param, float threshold_min_in_meter,
                           float threshold_max_in_meter)
{
    auto depth_intrin_ptr = param.get_depth_intrinsics_ptr();
    auto color_intrin_ptr = param.get_color_intrinsics_ptr();
    auto depth2color_extrin_ptr = param.get_depth2color_extrinsics_ptr();
    auto width = param.get_width();
    auto height = param.get_height();
    auto depth_scale = param.get_depth_scale();

    if (!depth_intrin_ptr || !color_intrin_ptr || !depth2color_extrin_ptr || !width || !height)
        return false;

    if (depth_scale - 0.0 < 0.0001)
        return false;

    auto depth_pixel_count = width * height;
    auto result_byte_size = sizeof(gca::point_t) * depth_pixel_count;

    std::shared_ptr<gca::point_t> result_ptr;
    if (!alloc_device(result_ptr, result_byte_size))
        return false;

    dim3 threads(32, 32);
    dim3 depth_blocks(div_up(width, threads.x), div_up(height, threads.y));

    __kernel_make_pointcloud_Z16_BGR8<<<depth_blocks, threads>>>(
        result_ptr.get(), width, height, cuda_depth_container.data(), cuda_color_container.data(),
        depth_intrin_ptr, color_intrin_ptr, depth2color_extrin_ptr, depth_scale,
        threshold_min_in_meter, threshold_max_in_meter);

    if (hipDeviceSynchronize() != hipSuccess)
        return false;

    hipMemcpy(result.data(), result_ptr.get(), result_byte_size, hipMemcpyDefault);

    return true;
}

/************************** This overload is used in point cloud class **************************/
bool cuda_make_point_cloud(thrust::device_vector<gca::point_t> &result,
                           const gca::cuda_depth_frame &cuda_depth_container,
                           const gca::cuda_color_frame &cuda_color_container,
                           const gca::cuda_camera_param &param, float threshold_min_in_meter,
                           float threshold_max_in_meter)
{
    auto depth_intrin_ptr = param.get_depth_intrinsics_ptr();
    auto color_intrin_ptr = param.get_color_intrinsics_ptr();
    auto depth2color_extrin_ptr = param.get_depth2color_extrinsics_ptr();
    auto width = param.get_width();
    auto height = param.get_height();
    auto depth_scale = param.get_depth_scale();

    if (!depth_intrin_ptr || !color_intrin_ptr || !depth2color_extrin_ptr || !width || !height)
        return false;

    if (depth_scale - 0.0 < 0.0001)
        return false;

    auto depth_pixel_count = width * height;
    result.resize(depth_pixel_count);

    dim3 threads(32, 32);
    dim3 depth_blocks(div_up(width, threads.x), div_up(height, threads.y));

    __kernel_make_pointcloud_Z16_BGR8<<<depth_blocks, threads>>>(
        result.data().get(), width, height, cuda_depth_container.data(),
        cuda_color_container.data(), depth_intrin_ptr, color_intrin_ptr, depth2color_extrin_ptr,
        depth_scale, threshold_min_in_meter, threshold_max_in_meter, true);

    if (hipDeviceSynchronize() != ::hipSuccess)
        return false;

    remove_invalid_points(result);

    return true;
}

/****************** Voxel grid Downsampling with Eigen Vector3f as coordinates ******************/
/**************************** Useful functors for thrust algorithms  ****************************/

struct add_points_functor
{
    __device__ gca::point_t operator()(const gca::point_t &first, const gca::point_t &second)
    {

        return gca::point_t{.coordinates{
                                .x = first.coordinates.x + second.coordinates.x,
                                .y = first.coordinates.y + second.coordinates.y,
                                .z = first.coordinates.z + second.coordinates.z,
                            },
                            .r = first.r + second.r,
                            .g = first.g + second.g,
                            .b = first.b + second.b,
                            .property = gca::point_property::inactive};
    }
};

struct compute_points_mean_functor
{
    compute_points_mean_functor(const uint32_t min_points_n_threshold)
        : m_threshold(min_points_n_threshold)
    {
    }

    const uint32_t m_threshold;

    __device__ gca::point_t operator()(const gca::point_t &points_sum, const uint32_t n)
    {
        if (n <= m_threshold)
        {
            return gca::point_t{.property = gca::point_property::invalid};
        }

        return gca::point_t{.coordinates{
                                .x = points_sum.coordinates.x / n,
                                .y = points_sum.coordinates.y / n,
                                .z = points_sum.coordinates.z / n,
                            },
                            .r = points_sum.r / n,
                            .g = points_sum.g / n,
                            .b = points_sum.b / n,
                            .property = gca::point_property::inactive};
    }
};

/*********************************** Voxel grid down sampling ***********************************/

::hipError_t cuda_voxel_grid_downsample(thrust::device_vector<gca::point_t> &result_points,
                                         const thrust::device_vector<gca::point_t> &src_points,
                                         const float3 &voxel_grid_min_bound, const float voxel_size,
                                         const uint32_t min_points_num_in_one_voxel)
{
    auto n_points = src_points.size();
    if (result_points.size() != n_points)
    {
        return ::hipErrorInvalidValue;
    }

    thrust::device_vector<int3> keys(n_points);

    thrust::transform(src_points.begin(), src_points.end(), keys.begin(),
                      compute_voxel_key_functor(voxel_grid_min_bound, voxel_size));
    auto err = hipGetLastError();
    if (err != ::hipSuccess)
    {
        return err;
    }

    thrust::device_vector<gca::point_t> sorted_point_cloud(n_points);
    thrust::device_vector<size_t> index_vec(n_points);
    thrust::sequence(index_vec.begin(), index_vec.end());
    thrust::sort_by_key(keys.begin(), keys.end(), index_vec.begin(), compare_voxel_key_functor());
    thrust::gather(index_vec.begin(), index_vec.end(), src_points.begin(),
                   sorted_point_cloud.begin());
    err = hipGetLastError();
    if (err != ::hipSuccess)
    {
        return err;
    }

    /* The following line of sorting code is only for memory. RIP
       man.... By using it the sorting time is 5ms. But this is
       too slow for me. So i made a vector which contains only the
       index of the points and only let it be sorted. And than I
       tried to get every point by using the src_points vector and
       sorted index. This really works and the whole process runs
       less than 1.5 ms. I'm very happy about it!!! */
    // thrust::sort_by_key(keys.begin(), keys.end(),
    // sorted_point_cloud.begin(), compare_voxel_key_functor());

    thrust::device_vector<uint32_t> points_counter_per_voxel(n_points, 1);
    thrust::device_vector<uint32_t> result_points_counter(n_points);

    auto end_iter_of_points =
        thrust::reduce_by_key(keys.begin(), keys.end(), sorted_point_cloud.begin(),
                              thrust::make_discard_iterator(), result_points.begin(),
                              voxel_key_equal_functor(), add_points_functor())
            .second;
    err = hipGetLastError();
    if (err != ::hipSuccess)
    {
        return err;
    }

    auto end_iter_of_points_counter =
        thrust::reduce_by_key(keys.begin(), keys.end(), points_counter_per_voxel.begin(),
                              thrust::make_discard_iterator(), result_points_counter.begin(),
                              voxel_key_equal_functor())
            .second;
    err = hipGetLastError();
    if (err != ::hipSuccess)
    {
        return err;
    }

    auto new_n_points = end_iter_of_points - result_points.begin();
    if (new_n_points != (end_iter_of_points_counter - result_points_counter.begin()))
    {
        return ::hipErrorInvalidValue;
    }

    result_points.resize(new_n_points);
    result_points_counter.resize(new_n_points);

    thrust::transform(result_points.begin(), result_points.end(), result_points_counter.begin(),
                      result_points.begin(),
                      compute_points_mean_functor(min_points_num_in_one_voxel));
    if (err != ::hipSuccess)
    {
        return err;
    }

    remove_invalid_points(result_points);

    return ::hipSuccess;
}
} // namespace gca
