#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/gather.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

#include "camera/camera_param.hpp"
#include "cuda_container/cuda_container.hpp"
#include "geometry/cuda_point_cloud_factory.cuh"
#include "geometry/geometry_util.cuh"
#include "geometry/type.hpp"
#include "util/cuda_util.cuh"

namespace gca
{
/************************************ CUDA bilateral_filter  ************************************/

__forceinline__ __device__ static float __gaussian(float x, float sigma)
{
    return exp(-(x * x) / (2 * sigma * sigma));
}

__device__ static float __bilateral_filter(const uint16_t *input, uint32_t input_width,
                                           uint32_t input_height, int index_x, int index_y,
                                           int filter_radius, float sigma_space, float sigma_depth)
{
    float sum_weight = 0.0f;
    float sum = 0.0f;

#pragma unroll
    for (int dy = -filter_radius; dy <= filter_radius; ++dy)
    {
#pragma unroll
        for (int dx = -filter_radius; dx <= filter_radius; ++dx)
        {
            int nx = index_x + dx;
            int ny = index_y + dy;

            if (nx >= 0 && nx < input_width && ny >= 0 && ny < input_height)
            {
                float weight = __gaussian(sqrtf(dx * dx + dy * dy), sigma_space) *
                               __gaussian(abs(input[index_y * input_width + index_x] -
                                              input[ny * input_width + nx]),
                                          sigma_depth);

                sum_weight += weight;
                sum += weight * input[ny * input_width + nx];
            }
        }
    }

    return sum / sum_weight;
}

/****************** Create point cloud from rgbd, include invalid point remove ******************/

__forceinline__ __device__ static void __transform_point_to_point(float to_point[3],
                                                                  const float from_point[3],
                                                                  const gca::extrinsics &extrin)
{
    to_point[0] = extrin.rotation[0] * from_point[0] + extrin.rotation[3] * from_point[1] +
                  extrin.rotation[6] * from_point[2] + extrin.translation[0];
    to_point[1] = extrin.rotation[1] * from_point[0] + extrin.rotation[4] * from_point[1] +
                  extrin.rotation[7] * from_point[2] + extrin.translation[1];
    to_point[2] = extrin.rotation[2] * from_point[0] + extrin.rotation[5] * from_point[1] +
                  extrin.rotation[8] * from_point[2] + extrin.translation[2];
}

__forceinline__ __device__ static void __depth_uv_to_xyz(const float uv[2], const float depth,
                                                         float xyz[3],
                                                         const gca::intrinsics &depth_intrin)
{
    auto z = depth;
    xyz[2] = z;
    xyz[0] = (uv[0] - depth_intrin.cx) * z / depth_intrin.fx;
    xyz[1] = (uv[1] - depth_intrin.cy) * z / depth_intrin.fy;
}

__forceinline__ __device__ static void __xyz_to_color_uv(const float xyz[3], float uv[2],
                                                         const gca::intrinsics &color_intrin)
{
    uv[0] = (xyz[0] * color_intrin.fx / xyz[2]) + color_intrin.cx;
    uv[1] = (xyz[1] * color_intrin.fy / xyz[2]) + color_intrin.cy;
}

__global__ static void __kernel_make_pointcloud_Z16_BGR8(
    gca::point_t *point_set_out, const uint32_t width, const uint32_t height,
    const uint16_t *depth_frame_data, const uint8_t *color_frame_data,
    const gca::intrinsics *depth_intrin_ptr, const gca::intrinsics *color_intrin_ptr,
    const gca::extrinsics *depth_to_color_extrin_ptr, const float depth_scale, float threshold_min,
    float threshold_max, bool if_bilateral_filter = false)
{
    __shared__ gca::intrinsics depth_intrin_shared;
    __shared__ gca::intrinsics color_intrin_shared;
    __shared__ gca::extrinsics depth_to_color_extrin_shared;

    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        depth_intrin_shared = *depth_intrin_ptr;
        color_intrin_shared = *color_intrin_ptr;
        depth_to_color_extrin_shared = *depth_to_color_extrin_ptr;
    }

    __syncthreads();

    int depth_x = blockIdx.x * blockDim.x + threadIdx.x;
    int depth_y = blockIdx.y * blockDim.y + threadIdx.y;
    int depth_pixel_index = depth_y * width + depth_x;

    // Shared memory or texture memory loading of depth_frame_data and color_frame_data

    if (depth_x >= 0 && depth_x < width && depth_y >= 0 && depth_y < height)
    {
        float depth_value;
        // Extract depth value
        if (if_bilateral_filter)
            depth_value =
                __bilateral_filter(depth_frame_data, width, height, depth_x, depth_y, 7, 1, 50) *
                depth_scale;
        else
            depth_value = depth_frame_data[depth_pixel_index] * depth_scale;

        if (depth_value <= 0.0 || depth_value < threshold_min || depth_value > threshold_max)
        {
            point_set_out[depth_pixel_index].property = gca::point_property::invalid;
            return;
        }

        // Calculate depth_uv and depth_xyz
        float depth_uv[2] = {depth_x - 0.5f, depth_y - 0.5f};
        float depth_xyz[3];
        __depth_uv_to_xyz(depth_uv, depth_value, depth_xyz, depth_intrin_shared);

        // Calculate color_xyz
        float color_xyz[3];
        __transform_point_to_point(color_xyz, depth_xyz, depth_to_color_extrin_shared);

        // Calculate color_uv
        float color_uv[2];
        __xyz_to_color_uv(color_xyz, color_uv, color_intrin_shared);

        const int target_x = static_cast<int>(color_uv[0] + 0.5f);
        const int target_y = static_cast<int>(color_uv[1] + 0.5f);

        if (target_x >= 0 && target_x < width && target_y >= 0 && target_y < height)
        {
            gca::point_t p;

            p.coordinates.x = depth_xyz[0];
            p.coordinates.y = depth_xyz[1];
            p.coordinates.z = depth_xyz[2];

            const int color_index = 3 * (target_y * width + target_x);
            p.b = color_frame_data[color_index + 0];
            p.g = color_frame_data[color_index + 1];
            p.r = color_frame_data[color_index + 2];

            p.property = gca::point_property::inactive;

            point_set_out[depth_pixel_index] = p;
        }
        else
        {
            point_set_out[depth_pixel_index].property = gca::point_property::invalid;
            return;
        }
    }
}

/************************* For Debug, not be used in point cloud class **************************/
bool cuda_make_point_cloud(std::vector<gca::point_t> &result,
                           const gca::cuda_depth_frame &cuda_depth_container,
                           const gca::cuda_color_frame &cuda_color_container,
                           const gca::cuda_camera_param &param, float threshold_min_in_meter,
                           float threshold_max_in_meter)
{
    auto depth_intrin_ptr = param.get_depth_intrinsics_ptr();
    auto color_intrin_ptr = param.get_color_intrinsics_ptr();
    auto depth2color_extrin_ptr = param.get_depth2color_extrinsics_ptr();
    auto width = param.get_width();
    auto height = param.get_height();
    auto depth_scale = param.get_depth_scale();

    if (!depth_intrin_ptr || !color_intrin_ptr || !depth2color_extrin_ptr || !width || !height)
        return false;

    if (depth_scale - 0.0 < 0.0001)
        return false;

    auto depth_pixel_count = width * height;
    auto result_byte_size = sizeof(gca::point_t) * depth_pixel_count;

    std::shared_ptr<gca::point_t> result_ptr;
    if (!alloc_device(result_ptr, result_byte_size))
        return false;

    dim3 threads(32, 32);
    dim3 depth_blocks(div_up(width, threads.x), div_up(height, threads.y));

    __kernel_make_pointcloud_Z16_BGR8<<<depth_blocks, threads>>>(
        result_ptr.get(), width, height, cuda_depth_container.data(), cuda_color_container.data(),
        depth_intrin_ptr, color_intrin_ptr, depth2color_extrin_ptr, depth_scale,
        threshold_min_in_meter, threshold_max_in_meter);

    if (hipDeviceSynchronize() != hipSuccess)
        return false;

    hipMemcpy(result.data(), result_ptr.get(), result_byte_size, hipMemcpyDefault);

    return true;
}

/************************** This overload is used in point cloud class **************************/
bool cuda_make_point_cloud(thrust::device_vector<gca::point_t> &result,
                           const gca::cuda_depth_frame &cuda_depth_container,
                           const gca::cuda_color_frame &cuda_color_container,
                           const gca::cuda_camera_param &param, float threshold_min_in_meter,
                           float threshold_max_in_meter)
{
    auto depth_intrin_ptr = param.get_depth_intrinsics_ptr();
    auto color_intrin_ptr = param.get_color_intrinsics_ptr();
    auto depth2color_extrin_ptr = param.get_depth2color_extrinsics_ptr();
    auto width = param.get_width();
    auto height = param.get_height();
    auto depth_scale = param.get_depth_scale();

    if (!depth_intrin_ptr || !color_intrin_ptr || !depth2color_extrin_ptr || !width || !height)
        return false;

    if (depth_scale - 0.0 < 0.0001)
        return false;

    auto depth_pixel_count = width * height;
    result.resize(depth_pixel_count);

    dim3 threads(32, 32);
    dim3 depth_blocks(div_up(width, threads.x), div_up(height, threads.y));

    __kernel_make_pointcloud_Z16_BGR8<<<depth_blocks, threads>>>(
        result.data().get(), width, height, cuda_depth_container.data(),
        cuda_color_container.data(), depth_intrin_ptr, color_intrin_ptr, depth2color_extrin_ptr,
        depth_scale, threshold_min_in_meter,
        threshold_max_in_meter); // didnt use bilateral filter, later maybe a compare to see if it
                                 // is needed

    if (hipDeviceSynchronize() != ::hipSuccess)
        return false;

    remove_invalid_points(result);

    return true;
}

/****************** Voxel grid Downsampling with Eigen Vector3f as coordinates ******************/
/**************************** Useful functors for thrust algorithms  ****************************/

struct compute_voxel_key_functor
{
    compute_voxel_key_functor(const float3 &voxel_grid_min_bound, const float voxel_size)
        : m_voxel_grid_min_bound(voxel_grid_min_bound)
        , m_voxel_size(voxel_size)
    {
    }

    const float3 m_voxel_grid_min_bound;
    const float m_voxel_size;

    __forceinline__ __device__ int3 operator()(const gca::point_t &point)
    {
        int3 ref_coord;
        ref_coord.x =
            __float2int_rd((point.coordinates.x - m_voxel_grid_min_bound.x) / m_voxel_size);
        ref_coord.y =
            __float2int_rd((point.coordinates.y - m_voxel_grid_min_bound.y) / m_voxel_size);
        ref_coord.z =
            __float2int_rd((point.coordinates.z - m_voxel_grid_min_bound.z) / m_voxel_size);
        return ref_coord;
    }
};

struct compare_voxel_key_functor : public thrust::binary_function<int3, int3, bool>
{
    __forceinline__ __host__ __device__ bool operator()(const int3 &lhs, const int3 &rhs) const
    {
        if (lhs.x != rhs.x)
            return lhs.x < rhs.x;

        else if (lhs.y != rhs.y)
            return lhs.y < rhs.y;

        else if (lhs.z != rhs.z)
            return lhs.z < rhs.z;

        return false;
    }
};

struct voxel_key_equal_functor : public thrust::binary_function<int3, int3, bool>
{
    __forceinline__ __host__ __device__ bool operator()(const int3 &lhs, const int3 &rhs) const
    {
        return lhs.x == rhs.x && lhs.y == rhs.y && lhs.z == rhs.z;
    }
};

struct add_points_functor
{
    __forceinline__ __device__ gca::point_t operator()(const gca::point_t &first,
                                                       const gca::point_t &second)
    {

        return gca::point_t{.coordinates{
                                .x = first.coordinates.x + second.coordinates.x,
                                .y = first.coordinates.y + second.coordinates.y,
                                .z = first.coordinates.z + second.coordinates.z,
                            },
                            .r = first.r + second.r,
                            .g = first.g + second.g,
                            .b = first.b + second.b,
                            .property = gca::point_property::inactive};
    }
};

struct compute_points_mean_functor
{
    __forceinline__ __device__ gca::point_t operator()(const gca::point_t &points_sum,
                                                       const gca::counter_t n)
    {
        return gca::point_t{.coordinates{
                                .x = points_sum.coordinates.x / n,
                                .y = points_sum.coordinates.y / n,
                                .z = points_sum.coordinates.z / n,
                            },
                            .r = points_sum.r / n,
                            .g = points_sum.g / n,
                            .b = points_sum.b / n,
                            .property = gca::point_property::inactive};
    }
};

/*********************************** Voxel grid down sampling ***********************************/

::hipError_t cuda_voxel_grid_downsample(thrust::device_vector<gca::point_t> &result_points,
                                         const thrust::device_vector<gca::point_t> &src_points,
                                         const float3 &voxel_grid_min_bound, const float voxel_size)
{
    auto n_points = src_points.size();
    if (result_points.size() != n_points)
    {
        return ::hipErrorInvalidValue;
    }

    thrust::device_vector<int3> keys(n_points);
    thrust::transform(src_points.begin(), src_points.end(), keys.begin(),
                      compute_voxel_key_functor(voxel_grid_min_bound, voxel_size));
    auto err = hipGetLastError();
    if (err != ::hipSuccess)
    {
        return err;
    }

    thrust::device_vector<gca::index_t> index_vec(n_points);
    thrust::sequence(index_vec.begin(), index_vec.end());
    thrust::sort_by_key(keys.begin(), keys.end(), index_vec.begin(), compare_voxel_key_functor());
    auto get_point_with_sorted_index_iter =
        thrust::make_permutation_iterator(src_points.begin(), index_vec.begin());
    err = hipGetLastError();
    if (err != ::hipSuccess)
    {
        return err;
    }

    auto end_iter_of_points =
        thrust::reduce_by_key(keys.begin(), keys.end(), get_point_with_sorted_index_iter,
                              thrust::make_discard_iterator(), result_points.begin(),
                              voxel_key_equal_functor(), add_points_functor())
            .second;
    err = hipGetLastError();
    if (err != ::hipSuccess)
    {
        return err;
    }

    thrust::device_vector<gca::counter_t> points_counter_per_voxel(n_points, 1);
    auto end_iter_of_points_counter =
        thrust::reduce_by_key(keys.begin(), keys.end(), points_counter_per_voxel.begin(),
                              thrust::make_discard_iterator(), points_counter_per_voxel.begin(),
                              voxel_key_equal_functor())
            .second;
    err = hipGetLastError();
    if (err != ::hipSuccess)
    {
        return err;
    }

    auto new_n_points = end_iter_of_points - result_points.begin();
    if (new_n_points != (end_iter_of_points_counter - points_counter_per_voxel.begin()))
    {
        return ::hipErrorInvalidValue;
    }

    result_points.resize(new_n_points);
    points_counter_per_voxel.resize(new_n_points);

    thrust::transform(result_points.begin(), result_points.end(), points_counter_per_voxel.begin(),
                      result_points.begin(), compute_points_mean_functor());
    if (err != ::hipSuccess)
    {
        return err;
    }

    return ::hipSuccess;
}
} // namespace gca
