#include "hip/hip_runtime.h"
#include "cuda_color_icp_build_least_square.cuh"

#include "util/math.cuh"

#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/pair.h>
#include <thrust/tuple.h>

namespace gca
{
struct compute_residual_pair_functor
{
    compute_residual_pair_functor(const thrust::device_vector<gca::point_t> &tgt_points,
                                  const thrust::device_vector<float3> &tgt_normals,
                                  const thrust::device_vector<float3> &tgt_color_gradient,
                                  const float lambda)
        : m_tgt_points_ptr(thrust::raw_pointer_cast(tgt_points.data()))
        , m_tgt_normals_ptr(thrust::raw_pointer_cast(tgt_normals.data()))
        , m_tgt_color_gradient_ptr(thrust::raw_pointer_cast(tgt_color_gradient.data()))
        , m_sqrt_lambda(sqrtf(lambda))
        , m_sqrt_1_minus_lambda(sqrtf(1.0f - lambda))
    {
    }

    const gca::point_t *m_tgt_points_ptr;
    const float3 *m_tgt_normals_ptr;
    const float3 *m_tgt_color_gradient_ptr;
    const float m_sqrt_lambda;
    const float m_sqrt_1_minus_lambda;

    __forceinline__ __device__ thrust::pair<float, float> operator()(
        thrust::tuple<gca::point_t, gca::index_t> &pts_and_nn)
    {
        auto pts_src = thrust::get<0>(pts_and_nn);
        auto nn_idx_tgt = thrust::get<1>(pts_and_nn);

        auto nn_pts_tgt = m_tgt_points_ptr[nn_idx_tgt];
        auto normal = m_tgt_normals_ptr[nn_idx_tgt];
        auto color_gradient_tgt = m_tgt_color_gradient_ptr[nn_idx_tgt];

        auto r_ = dot((pts_src.coordinates - nn_pts_tgt.coordinates), normal);
        auto rg = m_sqrt_lambda * r_; // geometric

        auto indensity_pts_src = pts_src.color.to_intensity();
        auto indensity_pts_tgt = nn_pts_tgt.color.to_intensity();
        // project src point onto the tangent plane of taget point
        auto proj_coordinates = pts_src.coordinates - r_ * normal;

        auto indensity_proj = dot(color_gradient_tgt, (proj_coordinates - nn_pts_tgt.coordinates));

        auto rc = m_sqrt_1_minus_lambda * (indensity_pts_src - indensity_proj);

        return thrust::make_pair(rg, rc);
    }
};

::hipError_t cuda_compute_residual(
    thrust::device_vector<thrust::pair<float, float>> &result_rg_rc_pair,
    const thrust::device_vector<gca::point_t> &src_points,
    const thrust::device_vector<gca::point_t> &tgt_points,
    const thrust::device_vector<float3> &tgt_normals,
    const thrust::device_vector<float3> &tgt_color_gradient,
    const thrust::device_vector<gca::index_t> &nn_src_tgt, const float lambda)
{
    if (lambda > 1.0f)
    {
        return ::hipErrorInvalidValue;
    }

    auto n_points_src = src_points.size();
    if (nn_src_tgt.size() != n_points_src)
    {
        return ::hipErrorInvalidValue;
    }

    if (tgt_points.size() != tgt_normals.size() || tgt_points.size() != tgt_color_gradient.size())
    {
        return ::hipErrorInvalidValue;
    }

    if (result_rg_rc_pair.size() != n_points_src)
    {
        result_rg_rc_pair.resize(n_points_src);
    }

    auto zipped_begin =
        thrust::make_zip_iterator(thrust::make_tuple(src_points.begin(), nn_src_tgt.begin()));

    auto zipped_end =
        thrust::make_zip_iterator(thrust::make_tuple(src_points.end(), nn_src_tgt.end()));

    thrust::transform(
        zipped_begin, zipped_end, result_rg_rc_pair.begin(),
        compute_residual_pair_functor(tgt_points, tgt_normals, tgt_color_gradient, lambda));
    auto err = hipGetLastError();
    return err;
}
} // namespace gca
