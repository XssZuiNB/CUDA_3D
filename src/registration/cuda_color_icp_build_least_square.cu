#include "hip/hip_runtime.h"
#include "cuda_color_icp_build_least_square.cuh"

#include "util/math.cuh"

#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/pair.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>

namespace gca
{
struct compute_residual_pair_functor
{
    compute_residual_pair_functor(const thrust::device_vector<gca::point_t> &tgt_points,
                                  const thrust::device_vector<float3> &tgt_normals,
                                  const thrust::device_vector<float3> &tgt_color_gradient,
                                  const float lambda)
        : m_tgt_points_ptr(thrust::raw_pointer_cast(tgt_points.data()))
        , m_tgt_normals_ptr(thrust::raw_pointer_cast(tgt_normals.data()))
        , m_tgt_color_gradient_ptr(thrust::raw_pointer_cast(tgt_color_gradient.data()))
        , m_sqrt_lambda_geometry(sqrtf(lambda))
        , m_sqrt_lambda_color(sqrtf(1.0f - lambda))
    {
    }

    const gca::point_t *m_tgt_points_ptr;
    const float3 *m_tgt_normals_ptr;
    const float3 *m_tgt_color_gradient_ptr;
    const float m_sqrt_lambda_geometry;
    const float m_sqrt_lambda_color;

    __forceinline__ __device__ thrust::pair<float, float> operator()(
        const thrust::tuple<gca::point_t, gca::index_t> &pts_and_nn) const
    {
        auto pts_src = thrust::get<0>(pts_and_nn);
        auto nn_idx_tgt = thrust::get<1>(pts_and_nn);

        if (nn_idx_tgt < 0)
        {
            return thrust::make_pair(0.0f, 0.0f);
        }

        auto nn_pts_tgt = m_tgt_points_ptr[nn_idx_tgt];
        auto normal = m_tgt_normals_ptr[nn_idx_tgt];
        auto color_gradient_tgt = m_tgt_color_gradient_ptr[nn_idx_tgt];

        auto rg_ = dot((pts_src.coordinates - nn_pts_tgt.coordinates), normal);
        auto rg_weighted = m_sqrt_lambda_geometry * rg_; // geometric

        auto indensity_pts_src = pts_src.color.to_intensity();
        auto indensity_pts_tgt = nn_pts_tgt.color.to_intensity();
        // project src point onto the tangent plane of taget point
        auto proj_coordinates = pts_src.coordinates - rg_ * normal;
        // get projected intensity
        auto indensity_proj = indensity_pts_tgt +
                              dot(color_gradient_tgt, (proj_coordinates - nn_pts_tgt.coordinates));
        auto rc_weighted = m_sqrt_lambda_color * (indensity_proj - indensity_pts_src); // color

        return thrust::make_pair(rg_weighted, rc_weighted);
    }
};

struct compute_residual_functor : compute_residual_pair_functor
{
    compute_residual_functor(const thrust::device_vector<gca::point_t> &tgt_points,
                             const thrust::device_vector<float3> &tgt_normals,
                             const thrust::device_vector<float3> &tgt_color_gradient,
                             const float lambda)
        : compute_residual_pair_functor(tgt_points, tgt_normals, tgt_color_gradient, lambda)
    {
    }

    __forceinline__ __device__ float operator()(
        const thrust::tuple<gca::point_t, gca::index_t> &pts_and_nn) const
    {
        const auto &pair = compute_residual_pair_functor::operator()(pts_and_nn);
        return pair.first + pair.second;
    }
};

struct compute_rmse_functor
{
    compute_rmse_functor(const thrust::device_vector<gca::point_t> &tgt_points,
                         const thrust::device_vector<float3> &tgt_normals,
                         const thrust::device_vector<float3> &tgt_color_gradient,
                         const float lambda)
        : m_tgt_points_ptr(thrust::raw_pointer_cast(tgt_points.data()))
        , m_tgt_normals_ptr(thrust::raw_pointer_cast(tgt_normals.data()))
        , m_tgt_color_gradient_ptr(thrust::raw_pointer_cast(tgt_color_gradient.data()))
        , m_lambda_geometry(lambda)
        , m_lambda_color(1.0f - lambda)
    {
    }

    const gca::point_t *m_tgt_points_ptr;
    const float3 *m_tgt_normals_ptr;
    const float3 *m_tgt_color_gradient_ptr;
    const float m_lambda_geometry;
    const float m_lambda_color;

    __forceinline__ __device__ float operator()(
        const thrust::tuple<gca::point_t, gca::index_t> &pts_and_nn) const
    {
        auto pts_src = thrust::get<0>(pts_and_nn);
        auto nn_idx_tgt = thrust::get<1>(pts_and_nn);

        if (nn_idx_tgt < 0)
        {
            return 0.0f;
        }

        auto nn_pts_tgt = m_tgt_points_ptr[nn_idx_tgt];
        auto normal = m_tgt_normals_ptr[nn_idx_tgt];
        auto color_gradient_tgt = m_tgt_color_gradient_ptr[nn_idx_tgt];

        auto rg_ = dot((pts_src.coordinates - nn_pts_tgt.coordinates), normal);
        auto rg_square_weighted = m_lambda_geometry * rg_ * rg_;

        auto indensity_pts_src = pts_src.color.to_intensity();
        auto indensity_pts_tgt = nn_pts_tgt.color.to_intensity();
        // project src point onto the tangent plane of taget point
        auto proj_coordinates = pts_src.coordinates - rg_ * normal;
        // get projected intensity
        auto indensity_proj = indensity_pts_tgt +
                              dot(color_gradient_tgt, (proj_coordinates - nn_pts_tgt.coordinates));
        auto rc_ = indensity_proj - indensity_pts_src;
        auto rc_square_weighted = m_lambda_color * rc_ * rc_;

        return rg_square_weighted + rc_square_weighted;
    }
};

struct compute_JTJ_and_JTr_functor
{
    compute_JTJ_and_JTr_functor(const thrust::device_vector<gca::point_t> &tgt_points,
                                const thrust::device_vector<float3> &tgt_normals,
                                const thrust::device_vector<float3> &tgt_color_gradient,
                                const float lambda)
        : m_tgt_points_ptr(thrust::raw_pointer_cast(tgt_points.data()))
        , m_tgt_normals_ptr(thrust::raw_pointer_cast(tgt_normals.data()))
        , m_tgt_color_gradient_ptr(thrust::raw_pointer_cast(tgt_color_gradient.data()))
        , m_sqrt_lambda_geometry(sqrtf(lambda))
        , m_sqrt_lambda_color(sqrtf(1.0f - lambda))
    {
    }

    const gca::point_t *m_tgt_points_ptr;
    const float3 *m_tgt_normals_ptr;
    const float3 *m_tgt_color_gradient_ptr;
    const float m_sqrt_lambda_geometry;
    const float m_sqrt_lambda_color;

    __forceinline__ __device__ float operator()()
    {
    }

private:
    void compute_jacobian_and_residual(
        mat6x1 &J_geometry, mat6x1 &J_color, float &r_geometry, float &r_color,
        const thrust::tuple<gca::point_t, gca::index_t> &pts_and_nn) const
    {
        auto pts_src = thrust::get<0>(pts_and_nn);
        auto nn_idx_tgt = thrust::get<1>(pts_and_nn);

        if (nn_idx_tgt < 0)
        {
            J_geometry.set_zero();
            J_color.set_zero();
            r_geometry = 0;
            r_color = 0;
            return;
        }

        auto nn_pts_tgt = m_tgt_points_ptr[nn_idx_tgt];
        auto normal = m_tgt_normals_ptr[nn_idx_tgt];
        auto color_gradient_tgt = m_tgt_color_gradient_ptr[nn_idx_tgt];

        // r_geometry without weight, this is also needed in color part, so compute it here firstly.
        auto r_geometry_ = dot((pts_src.coordinates - nn_pts_tgt.coordinates), normal);

        // geometric
        // jacobian
        mat3x1 J_geo_rotation(m_sqrt_lambda_geometry * cross(pts_src.coordinates, normal));
        mat3x1 J_geo_tranlation(m_sqrt_lambda_geometry * normal);
        J_geometry.set_block<3, 1>(J_geo_rotation, 0, 0);
        J_geometry.set_block<3, 1>(J_geo_tranlation, 3, 0);

        // residual
        r_geometry = m_sqrt_lambda_geometry * r_geometry_;

        // color
        // jacobian
        // paper equation 28, 29
        mat3x1 normal_as_mat(normal);

        // J_f(s)
        mat3x3 df_div_ds(float3x3::get_identity());

        // from equation 9: df(s)/ds = I - n * nT
        df_div_ds = df_div_ds - normal_as_mat * normal_as_mat.get_transpose();

        // from equation 8: dC_p(u)/du = d_pT -> equation 29 = d_pT * (I - n * nT) * J_s(xi)
        mat3x1 color_gradient_tgt_as_mat(color_gradient_tgt);

        //  d_pT * (I - n * nT) is called here as dc_x_df
        float3 dc_x_df(color_gradient_tgt_as_mat.get_transpose() * df_div_ds);

        // derivation of the formula J_s(xi) see Masterthesis von Shengsi Xu
        mat3x1 J_color_fir(m_sqrt_lambda_color * cross(pts_src.coordinates, dc_x_df));
        mat3x1 J_color_sec(m_sqrt_lambda_color * dc_x_df);
        J_color.set_block<3, 1>(J_color_fir, 0, 0);
        J_color.set_block<3, 1>(J_color_sec, 0, 0);
        // residual
        auto indensity_pts_src = pts_src.color.to_intensity();
        auto indensity_pts_tgt = nn_pts_tgt.color.to_intensity();
        // project src point onto the tangent plane of taget point
        auto proj_coordinates = pts_src.coordinates - r_geometry_ * normal;
        // get projected intensity
        auto indensity_proj = indensity_pts_tgt +
                              dot(color_gradient_tgt, (proj_coordinates - nn_pts_tgt.coordinates));
        auto rc_weighted = m_sqrt_lambda_color * (indensity_proj - indensity_pts_src); // color
    }
};

::hipError_t cuda_compute_residual_color_icp(
    thrust::device_vector<thrust::pair<float, float>> &result_rg_rc_pair,
    const thrust::device_vector<gca::point_t> &src_points,
    const thrust::device_vector<gca::point_t> &tgt_points,
    const thrust::device_vector<float3> &tgt_normals,
    const thrust::device_vector<float3> &tgt_color_gradient,
    const thrust::device_vector<gca::index_t> &nn_src_tgt, const float lambda)
{
    if (lambda > 1.0f)
    {
        return ::hipErrorInvalidValue;
    }

    auto n_points_src = src_points.size();
    if (nn_src_tgt.size() != n_points_src)
    {
        return ::hipErrorInvalidValue;
    }

    if (tgt_points.size() != tgt_normals.size() || tgt_points.size() != tgt_color_gradient.size())
    {
        return ::hipErrorInvalidValue;
    }

    if (result_rg_rc_pair.size() != n_points_src)
    {
        result_rg_rc_pair.resize(n_points_src);
    }

    auto zipped_begin =
        thrust::make_zip_iterator(thrust::make_tuple(src_points.begin(), nn_src_tgt.begin()));

    auto zipped_end =
        thrust::make_zip_iterator(thrust::make_tuple(src_points.end(), nn_src_tgt.end()));

    thrust::transform(
        zipped_begin, zipped_end, result_rg_rc_pair.begin(),
        compute_residual_pair_functor(tgt_points, tgt_normals, tgt_color_gradient, lambda));

    return ::hipSuccess;
}

::hipError_t cuda_compute_residual_color_icp(
    thrust::device_vector<float> &result_rg_plus_rc,
    const thrust::device_vector<gca::point_t> &src_points,
    const thrust::device_vector<gca::point_t> &tgt_points,
    const thrust::device_vector<float3> &tgt_normals,
    const thrust::device_vector<float3> &tgt_color_gradient,
    const thrust::device_vector<gca::index_t> &nn_src_tgt, const float lambda)
{
    if (lambda > 1.0f)
    {
        return ::hipErrorInvalidValue;
    }

    auto n_points_src = src_points.size();
    if (nn_src_tgt.size() != n_points_src)
    {
        return ::hipErrorInvalidValue;
    }

    if (tgt_points.size() != tgt_normals.size() || tgt_points.size() != tgt_color_gradient.size())
    {
        return ::hipErrorInvalidValue;
    }

    if (result_rg_plus_rc.size() != n_points_src)
    {
        result_rg_plus_rc.resize(n_points_src);
    }

    auto zipped_begin =
        thrust::make_zip_iterator(thrust::make_tuple(src_points.begin(), nn_src_tgt.begin()));

    auto zipped_end =
        thrust::make_zip_iterator(thrust::make_tuple(src_points.end(), nn_src_tgt.end()));

    thrust::transform(
        zipped_begin, zipped_end, result_rg_plus_rc.begin(),
        compute_residual_functor(tgt_points, tgt_normals, tgt_color_gradient, lambda));

    return ::hipSuccess;
}

::hipError_t cuda_compute_rmse_color_icp(float &result_rmse,
                                          const thrust::device_vector<gca::point_t> &src_points,
                                          const thrust::device_vector<gca::point_t> &tgt_points,
                                          const thrust::device_vector<float3> &tgt_normals,
                                          const thrust::device_vector<float3> &tgt_color_gradient,
                                          const thrust::device_vector<gca::index_t> &nn_src_tgt,
                                          const float lambda)
{
    if (lambda > 1.0f)
    {
        return ::hipErrorInvalidValue;
    }

    auto n_points_src = src_points.size();
    if (nn_src_tgt.size() != n_points_src)
    {
        return ::hipErrorInvalidValue;
    }

    if (tgt_points.size() != tgt_normals.size() || tgt_points.size() != tgt_color_gradient.size())
    {
        return ::hipErrorInvalidValue;
    }

    auto zipped_begin =
        thrust::make_zip_iterator(thrust::make_tuple(src_points.begin(), nn_src_tgt.begin()));

    auto zipped_end =
        thrust::make_zip_iterator(thrust::make_tuple(src_points.end(), nn_src_tgt.end()));

    result_rmse = thrust::transform_reduce(
        zipped_begin, zipped_end,
        compute_rmse_functor(tgt_points, tgt_normals, tgt_color_gradient, lambda), 0.0f,
        thrust::plus<float>());

    return ::hipSuccess;
}

::hipError_t cuda_build_gauss_newton_color_icp(
    mat6x6 &JTJ, mat6x1 &JTr, const thrust::device_vector<gca::point_t> &src_points,
    const thrust::device_vector<gca::point_t> &tgt_points,
    const thrust::device_vector<float3> &tgt_normals,
    const thrust::device_vector<float3> &tgt_color_gradient,
    const thrust::device_vector<gca::index_t> &nn_src_tgt, const float lambda)
{
    if (lambda > 1.0f)
    {
        return ::hipErrorInvalidValue;
    }

    auto n_points_src = src_points.size();
    if (nn_src_tgt.size() != n_points_src)
    {
        return ::hipErrorInvalidValue;
    }

    if (tgt_points.size() != tgt_normals.size() || tgt_points.size() != tgt_color_gradient.size())
    {
        return ::hipErrorInvalidValue;
    }
}
} // namespace gca
