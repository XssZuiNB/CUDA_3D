#include "hip/hip_runtime.h"
#include "cuda_compute_color_gradient.cuh"

#include "geometry/cuda_nn_search.cuh"
#include "util/math.cuh"

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

namespace gca
{
__forceinline__ __device__ void ldlt(const mat3x3 &A, mat3x3 &L, mat3x3 &D)
{
    L.set_identity();
    D.set_zero();

    D(0, 0) = A(0, 0);
    L(1, 0) = A(1, 0) / D(0, 0);
    L(2, 0) = A(2, 0) / D(0, 0);
    D(1, 1) = A(1, 1) - L(1, 0) * L(1, 0) * D(0, 0);
    L(2, 1) = (A(2, 1) - L(1, 0) * L(2, 0) * D(0, 0)) / D(1, 1);
    D(2, 2) = A(2, 2) - L(2, 0) * L(2, 0) * D(0, 0) - L(2, 1) * L(2, 1) * D(1, 1);
}

// Ly = b
__forceinline__ __device__ mat3x1 solve_lower(const mat3x3 &L, const mat3x1 &b)
{
    mat3x1 y;
    y(0) = b(0);
    y(1) = b(1) - L(1, 0) * y(0);
    y(2) = b(2) - L(2, 0) * y(0) - L(2, 1) * y(1);
    return y;
}

// Dz = y
__forceinline__ __device__ mat3x1 solve_diagonal(const mat3x3 &D, const mat3x1 &y)
{
    mat3x1 z;
    z(0) = y(0) / D(0, 0);
    z(1) = y(1) / D(1, 1);
    z(2) = y(2) / D(2, 2);
    return z;
}

// L^Tx = z
__forceinline__ __device__ mat3x1 solve_upper(const mat3x3 &L, const mat3x1 &z)
{
    mat3x1 x;
    x(2) = z(2);
    x(1) = z(1) - L(2, 1) * x(2);
    x(0) = z(0) - L(1, 0) * x(1) - L(2, 0) * x(2);
    return x;
}

struct compute_color_gradient_functor
{
    compute_color_gradient_functor(
        const thrust::device_vector<gca::point_t> &pts,
        const thrust::device_vector<float3> &normals,
        const thrust::device_vector<gca::index_t> &all_neighbors,
        const thrust::device_vector<thrust::pair<gca::index_t, gca::counter_t>>
            &pair_neighbors_begin_idx_and_count)
        : m_pts_ptr(thrust::raw_pointer_cast(pts.data()))
        , m_normals_ptr(thrust::raw_pointer_cast(normals.data()))
        , m_all_neighbors_ptr(thrust::raw_pointer_cast(all_neighbors.data()))
        , m_neighbors_begin_idx_and_count_ptr(
              thrust::raw_pointer_cast(pair_neighbors_begin_idx_and_count.data()))
    {
    }

    const gca::point_t *m_pts_ptr;
    const float3 *m_normals_ptr;
    const gca::index_t *m_all_neighbors_ptr;
    const thrust::pair<gca::index_t, gca::counter_t> *m_neighbors_begin_idx_and_count_ptr;

    __forceinline__ __device__ float3 operator()(gca::index_t idx) const
    {
        const auto &pts(m_pts_ptr[idx]);
        const auto &normal(m_normals_ptr[idx]);
        const auto begin_idx(__ldg(&(m_neighbors_begin_idx_and_count_ptr[idx].first)));
        const auto knn(__ldg(&(m_neighbors_begin_idx_and_count_ptr[idx].second)));

        if (knn < 5)
        {
            return make_float3(0.0f, 0.0f, 0.0f);
        }

        float intensity = pts.color.to_intensity();

        mat3x3 AtA;
        mat3x1 Atb;
        AtA.set_zero();
        Atb.set_zero();

        /* equation 10 least square L(d_p)' = 0 => (f(p') - p)T * d_p = C(p') - C(P)
         * -> Ax = b => ATA * x = ATb
         * => (f(p') - p) * (f(p') - p)T * d_p = (f(p') - p) * (C(p') - C(P))
         * => d_p = ((f(p') - p) * (f(p') - p)T) ^ -1 * ((f(p') - p) * (C(p') - C(P))) */
        for (gca::index_t i = 0; i < knn; ++i)
        {
            const int nn_idx = __ldg(&m_all_neighbors_ptr[begin_idx + i]);
            // find itself, continue
            if (nn_idx == idx)
                continue;

            const auto &nn_pts = m_pts_ptr[nn_idx];
            const auto nn_pts_proj_coordinates =
                nn_pts.coordinates - dot(nn_pts.coordinates - pts.coordinates, normal) * normal;

            float nn_intensity = nn_pts.color.to_intensity();

            mat3x1 vec_pp_p(nn_pts_proj_coordinates - pts.coordinates);
            AtA += vec_pp_p * vec_pp_p.get_transpose();
            Atb += vec_pp_p * (nn_intensity - intensity);
        }

        const mat3x1 n_mat(normal);
        // orthogonal constraint  (after equation 10 in paper)
        // Goal is to give an addition term d_pT * n_p = 0 for least square to optimize
        // Because this color gradient should be on the tangent plane of the point
        // This term makes convergence faster!
        AtA += (knn - 1) * (knn - 1) * n_mat * n_mat.get_transpose();
        // Atb += (knn - 1) * n_mat; // This makes slower convergence...
        AtA(0, 0) += 1.0e-6f;
        AtA(1, 1) += 1.0e-6f;
        AtA(2, 2) += 1.0e-6f;

        mat3x3 L, D;
        ldlt(AtA, L, D);
        auto y = solve_lower(L, Atb);
        auto z = solve_diagonal(D, y);
        auto x = solve_upper(L, z);

        // inverse is numerically unstable, could cause problem sometimes.
        // const float3 x(AtA.get_inverse() * Atb);
        return x;
    }
};

::hipError_t cuda_compute_color_gradient(thrust::device_vector<float3> &result,
                                          const thrust::device_vector<gca::point_t> &pts,
                                          const thrust::device_vector<float3> &normals,
                                          const float3 min_bound, const float3 max_bound,
                                          const float search_radius)
{
    auto n_points = pts.size();
    if (n_points != normals.size())
    {
        return ::hipErrorInvalidValue;
    }

    thrust::device_vector<gca::index_t> all_neighbors;
    thrust::device_vector<thrust::pair<gca::index_t, gca::counter_t>>
        pair_neighbors_begin_idx_and_count;

    auto err = cuda_search_radius_neighbors(all_neighbors, pair_neighbors_begin_idx_and_count, pts,
                                            min_bound, max_bound, search_radius);
    if (err != ::hipSuccess)
    {
        return err;
    }

    if (result.size() != n_points)
    {
        result.resize(n_points);
    }

    auto func = compute_color_gradient_functor(pts, normals, all_neighbors,
                                               pair_neighbors_begin_idx_and_count);

    thrust::transform(thrust::make_counting_iterator<gca::index_t>(0),
                      thrust::make_counting_iterator<gca::index_t>(n_points), result.begin(), func);

    return ::hipSuccess;
}
} // namespace gca