#include "hip/hip_runtime.h"
#include "cuda_compute_color_gradient.cuh"

#include "geometry/cuda_nn_search.cuh"
#include "util/math.cuh"

#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

namespace gca
{
struct compute_color_gradient_functor
{
    compute_color_gradient_functor(
        const thrust::device_vector<gca::point_t> &pts,
        const thrust::device_vector<float3> &normals,
        const thrust::device_vector<gca::index_t> &all_neighbors,
        const thrust::device_vector<thrust::pair<gca::index_t, gca::counter_t>>
            &pair_neighbors_begin_idx_and_count)
        : m_pts_ptr(thrust::raw_pointer_cast(pts.data()))
        , m_normals_ptr(thrust::raw_pointer_cast(normals.data()))
        , m_all_neighbors_ptr(thrust::raw_pointer_cast(all_neighbors.data()))
        , m_neighbors_begin_idx_and_count_ptr(
              thrust::raw_pointer_cast(pair_neighbors_begin_idx_and_count.data()))
    {
    }

    const gca::point_t *m_pts_ptr;
    const float3 *m_normals_ptr;
    const gca::index_t *m_all_neighbors_ptr;
    const thrust::pair<gca::index_t, gca::counter_t> *m_neighbors_begin_idx_and_count_ptr;

    __forceinline__ __device__ float3 operator()(gca::index_t idx) const
    {
        const auto pts(m_pts_ptr[idx]);
        const auto normal(m_normals_ptr[idx]);
        const auto begin_idx(__ldg(&(m_neighbors_begin_idx_and_count_ptr[idx].first)));
        const auto knn(__ldg(&(m_neighbors_begin_idx_and_count_ptr[idx].second)));

        if (knn < 5)
        {
            return make_float3(0.0f, 0.0f, 0.0f);
        }

        float intensity = pts.color.to_intensity();

        mat3x3 AtA;
        mat3x1 Atb;
        AtA.set_zero();
        Atb.set_zero();

        for (gca::index_t i = 0; i < knn; ++i)
        {
            const int nn_idx = __ldg(&m_all_neighbors_ptr[begin_idx + i]);
            // find itself, continue
            if (nn_idx == idx)
                continue;

            const auto &nn_pts = m_pts_ptr[nn_idx];
            const auto p_proj_coordinates =
                nn_pts.coordinates - dot(nn_pts.coordinates - pts.coordinates, normal) * normal;

            float nn_intensity = nn_pts.color.to_intensity();

            const mat3x1 vec_pp_p(p_proj_coordinates - pts.coordinates);
            AtA += vec_pp_p * vec_pp_p.get_transpose();
            Atb += vec_pp_p * (nn_intensity - intensity);
        }
        // orthogonal constraint
        const mat3x1 n_mat(normal);

        AtA += (knn - 1) * (knn - 1) * n_mat * n_mat.get_transpose();
        AtA(0, 0) += 1.0e-6;
        AtA(1, 1) += 1.0e-6;
        AtA(2, 2) += 1.0e-6;

        const auto x(AtA.get_inverse() * Atb);
        return make_float3(x(0), x(1), x(2));
    }
};

::hipError_t cuda_compute_color_gradient(thrust::device_vector<float3> &result,
                                          const thrust::device_vector<gca::point_t> &pts,
                                          const thrust::device_vector<float3> &normals,
                                          const float3 min_bound, const float3 max_bound,
                                          const float search_radius)
{
    auto n_points = pts.size();
    if (n_points != normals.size())
    {
        return ::hipErrorInvalidValue;
    }

    thrust::device_vector<gca::index_t> all_neighbors;
    thrust::device_vector<thrust::pair<gca::index_t, gca::counter_t>>
        pair_neighbors_begin_idx_and_count;

    auto err = cuda_search_radius_neighbors(all_neighbors, pair_neighbors_begin_idx_and_count, pts,
                                            min_bound, max_bound, search_radius);
    if (err != ::hipSuccess)
    {
        return err;
    }

    if (result.size() != n_points)
    {
        result.resize(n_points);
    }

    auto func = compute_color_gradient_functor(pts, normals, all_neighbors,
                                               pair_neighbors_begin_idx_and_count);

    thrust::transform(thrust::make_counting_iterator<gca::index_t>(0),
                      thrust::make_counting_iterator<gca::index_t>(n_points), result.begin(), func);

    return ::hipSuccess;
}
} // namespace gca